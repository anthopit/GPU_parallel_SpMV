#include "hip/hip_runtime.h"

#include <iostream>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers

extern "C" {
#include "mmio.h"
#include "matrix_utils.h"
}

// Number of threads per block
#define BD 256

const dim3 BLOCK_DIM(BD);

void CpuMatrixVector(int m, int n, const int* irp, const int* ja, const double* as, const double* x, double* y) {
    for (int row = 0; row < m; ++row) {
        double t=0.0;
        for (int col = irp[row]; col < irp[row+1]; ++col) {
            t = t + as[col]*x[ja[col]];
        }
        y[row] = t;
    }
}

__device__ void rowReduce(volatile double *sdata, int tid) {
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid +  8];
    sdata[tid] += sdata[tid +  4];
    sdata[tid] += sdata[tid +  2];
    sdata[tid] += sdata[tid +  1];
}

__global__ void gpuMatrixVectorCSR(int rows, const int* irp, const int* ja, const double* as,
                                const double* x, double* y) {
    __shared__ double aux[BD];
    int tc     = threadIdx.x;
    int row    = blockIdx.x;
    aux[tc] = 0.0;
    double t  = 0.0;
    if (row < rows) {
        for (int ic=irp[row] + tc;  ic<irp[row+1]; ic += blockDim.x) {
            t += as[ic]*x[ja[ic]];
        }
    }
    aux[tc] = t;

    __syncthreads();
    for (int s=BD/2; s >=32; s >>=1){
        if (tc<s)
            aux[tc] += aux[tc+s];
        __syncthreads();
    }

    if (tc<16) rowReduce(aux,tc);

    if (tc == 0)
        y[row] = aux[tc];
}

int main(int argc, char** argv) {

    int ret_code;
    MM_typecode matcode;
    FILE *f;

    int m, n, nz;

    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s [martix-market-filename]\n", argv[0]);
        exit(1);
    }
    else
    {
        if ((f = fopen(argv[1], "r")) == NULL){
            printf("Could not open file %s", argv[1]);
            exit(1);
        }
    }

    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        exit(1);
    }

    /*  This is how one can screen matrix types if their application */
    /*  only supports a subset of the Matrix Market data types.      */

    if (mm_is_complex(matcode) && mm_is_matrix(matcode) &&
        mm_is_sparse(matcode) && mm_is_integer(matcode))
    {
        printf("Sorry, this application does not support ");
        printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
        exit(1);
    }

    /* find out size of sparse matrix .... */

    if ((ret_code = mm_read_mtx_crd_size(f, &m, &n, &nz)) !=0)
        exit(1);


    // ----------------------- Host memory initialisation ----------------------- //

    int *h_irp, *h_ja;
    double *h_as;
    double* h_x = new double[n];
    double* h_y = new double[m];
    double* h_y_d = new double[m];

    srand(123456);
    for (int col = 0; col < m; ++col) {
        h_x[col] = 100.0f * static_cast<double>(rand()) / RAND_MAX;
    }

    read_mtx_coo_csr(f, m, n, &nz, &h_irp, &h_ja, &h_as, mm_is_symmetric(matcode), mm_is_pattern(matcode));
    //print_csr_mtx_csr(m, n, nz, h_irp, h_ja, h_as);
    //print_csr_mtx_2D(m, n, nz, h_irp, h_ja, h_as);

    // ---------------------- Device memory initialisation ---------------------- //

    int *d_irp, *d_ja;
    double *d_as;
    double *d_x, *d_y;

    checkCudaErrors(hipMalloc((void**) &d_irp, (m+1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &d_ja, nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &d_as, nz * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_x, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_y, m * sizeof(double)));

    checkCudaErrors(hipMemcpy(d_irp, h_irp, (m+1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_ja, h_ja, nz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_as, h_as, nz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, h_x,  n * sizeof(double), hipMemcpyHostToDevice));

    // ------------------------ Calculations on the CPU ------------------------- //
    double flopcnt=2.e-6*nz;

    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);

    timer->start();
    CpuMatrixVector(m, n,h_irp,h_ja,h_as,h_x,h_y);

    timer->stop();
    double cpuflops=flopcnt/ timer->getTime();
    std::cout << "  CPU time: " << timer->getTime() << " ms." << " GFLOPS " << cpuflops << std::endl;

// ------------------------ Calculations on the GPU ------------------------- //

    // Set the grid dimension to get one block per row.
    const dim3 GRID_DIM(m,1);

    timer->reset();
    timer->start();
    gpuMatrixVectorCSR<<<GRID_DIM, BLOCK_DIM >>>(m, d_irp, d_ja, d_as, d_x, d_y);
    checkCudaErrors(hipDeviceSynchronize());

    timer->stop();
    double gpuflops=flopcnt/ timer->getTime();
    std::cout << "  GPU time: " << timer->getTime() << " ms." << " GFLOPS " << gpuflops<<std::endl;

    checkCudaErrors(hipMemcpy(h_y_d, d_y, m*sizeof(double),hipMemcpyDeviceToHost));

    double reldiff = 0.0f;
    double diff = 0.0f;

    for (int row = 0; row < m; ++row) {
        double maxabs = std::max(std::abs(h_y[row]),std::abs(h_y_d[row]));
        if (maxabs == 0.0) maxabs=1.0;
        reldiff = std::max(reldiff, std::abs(h_y[row] - h_y_d[row])/maxabs);
        diff = std::max(diff, std::abs(h_y[row] - h_y_d[row]));
        //std::cout << row<<" "<<h_y[row]<<" "<<h_y_d[row] <<std::endl;
    }
    std::cout << "Max diff = " << diff << "  Max rel diff = " << reldiff << std::endl;
    std::cout << "Block size = " << BLOCK_DIM.x << std::endl;

// ------------------------------- Cleaning up ------------------------------ //

    delete timer;

    checkCudaErrors(hipFree(d_irp));
    checkCudaErrors(hipFree(d_ja));
    checkCudaErrors(hipFree(d_as));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));

    delete[] h_irp;
    delete[] h_ja;
    delete[] h_as;
    delete[] h_x;
    delete[] h_y;
    delete[] h_y_d;
    return 0;
}