#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers

extern "C" {
#include "mmio.h"
#include "matrix_utils.h"
}

#define XBD 8
#define YBD 8
const dim3 BLOCK_DIM(XBD,YBD);

void convert2DArrTo1D(int m, int n, double** A, double* B) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            B[i*n + j] = A[i][j];
        }
    }
}

void convert2DArrTo1D(int m, int n, int** A, int* B) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            B[i*n + j] = A[i][j];
        }
    }
}

void CpuMatrixVector(int m, const int* maxnz, const int* ja, const double* as, const double* x, double* y) {
    int i,j;
    for (i = 0; i < m; ++i) {
        double t=0.0;
        for (j = 0; j < *maxnz; ++j) {
            t = t + as[i*(*maxnz)+j]*x[ja[i*(*maxnz)+j]];
        }
        y[i] = t;
    }
}

__device__ void rowReduce(volatile double *sdata, int tid, int s) {
  switch(s){
  case 16:  sdata[tid] += sdata[tid + 16];
  case  8:  sdata[tid] += sdata[tid +  8];
  case  4:  sdata[tid] += sdata[tid +  4];
  case  2:  sdata[tid] += sdata[tid +  2];
  case  1:  sdata[tid] += sdata[tid +  1];
  }
}

__global__ void gpuMatrixVectorELL(int rows, const int maxnz, const int* ja,
                                   const double* as, const double* x, double* y) {
  __shared__ double ax[YBD][XBD];
  int tr     = threadIdx.y;
  int tc     = threadIdx.x;
  int row    = blockIdx.x*blockDim.y + tr;
  int s;
  ax[tr][tc] = 0.0;
  if (row < rows) {
    // Starting address of indexing within matrix A
    int idxm = row*maxnz+tc;
    int ic   = tc;
    double t  = 0.0;
    for ( ; ic<maxnz; ic += XBD) {
        t += as[idxm]*x[ja[idxm]];
        idxm += XBD;
    }
    ax[tr][tc] = t;
  }
  __syncthreads();

  for (s=XBD/2; s >=32; s >>=1){
    if (tc<s)
      ax[tr][tc] += ax[tr][tc+s]; 
    __syncthreads();
  }

  s = min(16,XBD/2);
  if (tc < s) rowReduce(&(ax[tr][0]),tc,s);
  
  if ((tc == 0)&&(row<rows))
    y[row] = ax[tr][tc];
  
}

int main(int argc, char** argv) {

    int ret_code;
    MM_typecode matcode;
    FILE *f;

    int m, n, nz;

    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s [martix-market-filename]\n", argv[0]);
        exit(1);
    }
    else
    {
        if ((f = fopen(argv[1], "r")) == NULL){
            printf("Could not open file %s", argv[1]);
            exit(1);
        }
    }

    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        exit(1);
    }

    /*  This is how one can screen matrix types if their application */
    /*  only supports a subset of the Matrix Market data types.      */

    if (mm_is_complex(matcode) && mm_is_matrix(matcode) &&
        mm_is_sparse(matcode) && mm_is_integer(matcode))
    {
        printf("Sorry, this application does not support ");
        printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
        exit(1);
    }

    /* find out size of sparse matrix .... */

    if ((ret_code = mm_read_mtx_crd_size(f, &m, &n, &nz)) !=0)
        exit(1);
  
  
  // ----------------------- Host memory initialisation ----------------------- //

    int h_maxnz;
    int **ja;
    double **as;
    double* h_x = new double[n];
    double* h_y = new double[m];
    double* h_y_d = new double[m];

    srand(123456);
    for (int col = 0; col < m; ++col) {
        h_x[col] = 100.0 * static_cast<double>(rand()) / RAND_MAX;
    }

    read_mtx_coo_ellpack(f, m, n, &nz, &h_maxnz, &ja, &as, mm_is_symmetric(matcode), mm_is_pattern(matcode));
//   print_ellpack_mtx_ellpack(m, n, &maxnz, ja, as);
//   print_ellpack_mtx_2D(m, n, &maxnz, ja, as);

    int* h_ja;
    double* h_as;
    h_ja = (int *) calloc(m * h_maxnz, sizeof(int));
    h_as = (double *) calloc(m * h_maxnz, sizeof(double));
    convert2DArrTo1D(m, h_maxnz, ja, h_ja);
    convert2DArrTo1D(m, h_maxnz, as, h_as);

// ---------------------- Device memory initialisation ---------------------- //

    int *d_ja;
    double *d_as, *d_x, *d_y;

    checkCudaErrors(hipMalloc((void**) &d_ja, m * h_maxnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &d_as, m * h_maxnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_x, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_y, m * sizeof(double)));

    checkCudaErrors(hipMemcpy(d_ja, h_ja, m * h_maxnz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_as, h_as, m * h_maxnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, h_x,  n * sizeof(double), hipMemcpyHostToDevice));

  // ------------------------ Calculations on the CPU ------------------------- //
    double flopcnt=2.e-6*nz;

    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);

    timer->start();
    CpuMatrixVector(m, &h_maxnz,h_ja,h_as,h_x,h_y);

    timer->stop();
    double cpuflops=flopcnt/ timer->getTime();
    std::cout << "  CPU time: " << timer->getTime() << " ms." << " GFLOPS " << cpuflops << std::endl;
  
// ------------------------ Calculations on the GPU ------------------------- //

  const dim3 GRID_DIM((m - 1+ BLOCK_DIM.y)/ BLOCK_DIM.y  ,1);

  timer->reset();
  timer->start();
  gpuMatrixVectorELL<<<GRID_DIM, BLOCK_DIM >>>(m, h_maxnz, d_ja, d_as, d_x, d_y);
  hipDeviceSynchronize();

  timer->stop();
  double gpuflops=flopcnt/ timer->getTime();
  std::cout << "  GPU time: " << timer->getTime() << " ms." << " GFLOPS " << gpuflops<<std::endl;

  checkCudaErrors(hipMemcpy(h_y_d, d_y, m*sizeof(double),hipMemcpyDeviceToHost));

  double reldiff = 0.0;
  double diff = 0.0;
  
  for (int row = 0; row < m; ++row) {
    double maxabs = std::max(std::abs(h_y[row]),std::abs(h_y_d[row]));
    if (maxabs == 0.0) maxabs=1.0;
    reldiff = std::max(reldiff, std::abs(h_y[row] - h_y_d[row])/maxabs);
    diff = std::max(diff, std::abs(h_y[row] - h_y_d[row]));
  }
  std::cout << "Block size = " << BLOCK_DIM.x << "x" << BLOCK_DIM.y << std::endl;
  std::cout << "Grid size = " << GRID_DIM.x << "x" << GRID_DIM.y << std::endl;
  std::cout << "Max diff = " << diff << "  Max rel diff = " << reldiff << std::endl;


// ------------------------------- Cleaning up ------------------------------ //

  delete timer;

  checkCudaErrors(hipFree(d_ja));
  checkCudaErrors(hipFree(d_as));
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));

  delete[] h_ja;
  delete[] h_as;
  delete[] h_x;
  delete[] h_y;
  delete[] h_y_d;
  return 0;
}
