#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers

extern "C" {
#include "mmio.h"
#include "matrix_utils.h"
}

#define BD 32

const dim3 BLOCK_DIM(BD);

void convert2DArrTo1D(int m, int n, double** A, double* B) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            B[i*n + j] = A[i][j];
        }
    }
}

void convert2DArrTo1D(int m, int n, int** A, int* B) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            B[i*n + j] = A[i][j];
        }
    }
}


void CpuMatrixVector(int m, const int* maxnz, const int* ja, const double* as, const double* x, double* y) {
    int i,j;
    for (i = 0; i < m; ++i) {
        double t=0.0;
        for (j = 0; j < *maxnz; ++j) {
            t = t + as[i*(*maxnz)+j]*x[ja[i*(*maxnz)+j]];
        }
        y[i] = t;
    }
}


__device__ void rowReduce(volatile double *sdata, int tid) {
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid +  8];
    sdata[tid] += sdata[tid +  4];
    sdata[tid] += sdata[tid +  2];
    sdata[tid] += sdata[tid +  1];
}

__global__ void gpuMatrixVectorELL(const int rows, const int maxnz, const int* ja, const double* as,
                                   const double* x, double* y) {
    __shared__ double aux[BD];
    int tc     = threadIdx.x;
    int row    = blockIdx.x;
    aux[tc] = 0.0;

    if (row < rows) {
        // Starting address of indexing within matrix A
        int idxm = row*maxnz+tc;
        double t  = 0.0;
        for (int ic=tc;  ic<(maxnz); ic += blockDim.x) {
            t = t + as[idxm]*x[ja[idxm]];
            idxm +=  blockDim.x;
        }
        aux[tc] = t;
    }
    __syncthreads();

    for (int s=BD/2; s >=32; s >>=1){
        if (tc<s)
            aux[tc] += aux[tc+s];
        __syncthreads();
    }

    if (tc<16) rowReduce(aux,tc);

    if (tc == 0)
        y[row] = aux[tc];
}

int main(int argc, char** argv) {

    int ret_code;
    MM_typecode matcode;
    FILE *f;

    int m, n, nz;

    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s [martix-market-filename]\n", argv[0]);
        exit(1);
    }
    else
    {
        if ((f = fopen(argv[1], "r")) == NULL){
            printf("Could not open file %s", argv[1]);
            exit(1);
        }
    }

    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        exit(1);
    }

    /*  This is how one can screen matrix types if their application */
    /*  only supports a subset of the Matrix Market data types.      */

    if (mm_is_complex(matcode) && mm_is_matrix(matcode) &&
        mm_is_sparse(matcode) && mm_is_integer(matcode))
    {
        printf("Sorry, this application does not support ");
        printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
        exit(1);
    }

    /* find out size of sparse matrix .... */

    if ((ret_code = mm_read_mtx_crd_size(f, &m, &n, &nz)) !=0)
        exit(1);


    // ----------------------- Host memory initialisation ----------------------- //

    int h_maxnz;
    int **ja;
    double **as;
    double* h_x = new double[n];
    double* h_y = new double[m];
    double* h_y_d = new double[m];

    srand(123456);
    for (int col = 0; col < m; ++col) {
        h_x[col] = 100.0f * static_cast<double>(rand()) / RAND_MAX;
    }

    read_mtx_coo_ellpack(f, m, n, &nz, &h_maxnz, &ja, &as, mm_is_symmetric(matcode), mm_is_pattern(matcode));
//   print_ellpack_mtx_ellpack(m, n, &maxnz, ja, as);
//   print_ellpack_mtx_2D(m, n, &maxnz, ja, as);

    int* h_ja = new int[m*h_maxnz];
    double* h_as = new double[m*h_maxnz];
    convert2DArrTo1D(m, h_maxnz, ja, h_ja);
    convert2DArrTo1D(m, h_maxnz, as, h_as);

// ---------------------- Device memory initialisation ---------------------- //

    int *d_ja;
    double *d_as, *d_x, *d_y;

    checkCudaErrors(hipMalloc((void**) &d_ja, m * h_maxnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &d_as, m * h_maxnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_x, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_y, m * sizeof(double)));

    checkCudaErrors(hipMemcpy(d_ja, h_ja, m * h_maxnz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_as, h_as, m * h_maxnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, h_x,  n * sizeof(double), hipMemcpyHostToDevice));

    // ------------------------ Calculations on the CPU ------------------------- //
    double flopcnt=2.e-6*nz;

    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);

    timer->start();
    CpuMatrixVector(m, &h_maxnz,h_ja,h_as,h_x,h_y);

    timer->stop();
    double cpuflops=flopcnt/ timer->getTime();
    std::cout << "  CPU time: " << timer->getTime() << " ms." << " GFLOPS " << cpuflops << std::endl;

// ------------------------ Calculations on the GPU ------------------------- //

    const dim3 GRID_DIM(m,1);

    timer->reset();
    timer->start();
    gpuMatrixVectorELL<<<GRID_DIM, BLOCK_DIM >>>(m, h_maxnz, d_ja, d_as, d_x, d_y);
    checkCudaErrors(hipDeviceSynchronize());

    timer->stop();
    double gpuflops=flopcnt/ timer->getTime();
    std::cout << "  GPU time: " << timer->getTime() << " ms." << " GFLOPS " << gpuflops<<std::endl;

    checkCudaErrors(hipMemcpy(h_y_d, d_y, m*sizeof(double),hipMemcpyDeviceToHost));

    double reldiff = 0.0f;
    double diff = 0.0f;

    for (int row = 0; row < m; ++row) {
        double maxabs = std::max(std::abs(h_y[row]),std::abs(h_y_d[row]));
        if (maxabs == 0.0) maxabs=1.0;
        reldiff = std::max(reldiff, std::abs(h_y[row] - h_y_d[row])/maxabs);
        diff = std::max(diff, std::abs(h_y[row] - h_y_d[row]));
    }
    std::cout << "Max diff = " << diff << "  Max rel diff = " << reldiff << std::endl;
    std::cout << "Block size = " << BLOCK_DIM.x << std::endl;

// ------------------------------- Cleaning up ------------------------------ //

    delete timer;

    checkCudaErrors(hipFree(d_ja));
    checkCudaErrors(hipFree(d_as));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));

    delete[] h_ja;
    delete[] h_as;
    delete[] h_x;
    delete[] h_y;
    delete[] h_y_d;
    return 0;
}
