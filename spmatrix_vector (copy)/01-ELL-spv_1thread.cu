#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers

extern "C" {
#include "mmio.h"
#include "matrix_utils.h"
}

#define BD 256

const dim3 BLOCK_DIM(BD);

void CpuMatrixVector(int m, const int* maxnz, const int* ja, const double* as, const double* x, double* y) {
    int i,j;
    for (i = 0; i < m; ++i) {
        double t=0.0;
        for (j = 0; j < *maxnz; ++j) {
            t = t + as[i*(*maxnz)+j]*x[ja[i*(*maxnz)+j]];
        }
        y[i] = t;
    }
}

__global__ void gpuMatrixVectorELL(int rows, const int* maxnz, const int* ja, const double* as,
                                const double* x, double* y) {
    int tr     = threadIdx.x;
    int row    = blockIdx.x*blockDim.x + tr;
    if (row < rows) {
        double t  = 0.0;
        for (int col = 0; col < *maxnz; ++col) {
            t = t + as[col + row * (*maxnz)]*x[ja[col + row * (*maxnz)]];
        }
        y[row] = t;
    }
}

int main(int argc, char** argv) {

    int ret_code;
    MM_typecode matcode;
    FILE *f;

    int m, n, nz;

    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s [martix-market-filename]\n", argv[0]);
        exit(1);
    }
    else
    {
        if ((f = fopen(argv[1], "r")) == NULL){
            printf("Could not open file %s", argv[1]);
            exit(1);
        }
    }

    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        exit(1);
    }

    /*  This is how one can screen matrix types if their application */
    /*  only supports a subset of the Matrix Market data types.      */

    if (mm_is_complex(matcode) && mm_is_matrix(matcode) &&
        mm_is_sparse(matcode) && mm_is_integer(matcode))
    {
        printf("Sorry, this application does not support ");
        printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
        exit(1);
    }

    /* find out size of sparse matrix .... */

    if ((ret_code = mm_read_mtx_crd_size(f, &m, &n, &nz)) !=0)
        exit(1);
  
  
  // ----------------------- Host memory initialisation ----------------------- //

    int h_maxnz;
    int **ja;
    double **as;
    double* h_x = new double[n];
    double* h_y = new double[m];
    double* h_y_d = new double[m];

    srand(123456);
    for (int col = 0; col < m; ++col) {
        h_x[col] = 100.0f * static_cast<double>(rand()) / RAND_MAX;
    }

    read_mtx_coo_ellpack(f, m, n, &nz, &h_maxnz, &ja, &as, mm_is_symmetric(matcode), mm_is_pattern(matcode));
//   print_ellpack_mtx_ellpack(m, n, &maxnz, ja, as);
//   print_ellpack_mtx_2D(m, n, &maxnz, ja, as);

    int* h_ja = new int[m*h_maxnz];
    double* h_as = new double[m*h_maxnz];
    for (int row = 0; row < m; ++row) {
        for (int col = 0; col < h_maxnz; ++col) {
            h_ja[col + row * h_maxnz] = ja[row][col];
            h_as[col + row * h_maxnz] = as[row][col];
        }
    }

    // ---------------------- Device memory initialisation ---------------------- //

    int *d_maxnz, *d_ja;
    double *d_as, *d_x, *d_y;

    checkCudaErrors(hipMalloc((void**) &d_maxnz, sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &d_ja, m * h_maxnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &d_as, m * h_maxnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_x, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_y, m * sizeof(double)));

    checkCudaErrors(hipMemcpy(d_maxnz, &h_maxnz, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_ja, h_ja, m * h_maxnz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_as, h_as, m * h_maxnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, h_x,  n * sizeof(double), hipMemcpyHostToDevice));

  // ------------------------ Calculations on the CPU ------------------------- //
    double flopcnt=2.e-6*nz;

    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);

    timer->start();
    CpuMatrixVector(m, &h_maxnz,h_ja,h_as,h_x,h_y);

    timer->stop();
    double cpuflops=flopcnt/ timer->getTime();
    std::cout << "  CPU time: " << timer->getTime() << " ms." << " GFLOPS " << cpuflops << std::endl;

// ------------------------ Calculations on the GPU ------------------------- //

    const dim3 GRID_DIM((m - 1 + BLOCK_DIM.x)/ BLOCK_DIM.x  ,1);

    timer->reset();
    timer->start();
    gpuMatrixVectorELL<<<GRID_DIM, BLOCK_DIM >>>(m, d_maxnz, d_ja, d_as, d_x, d_y);
    checkCudaErrors(hipDeviceSynchronize());

    timer->stop();
    double gpuflops=flopcnt/ timer->getTime();
    std::cout << "  GPU time: " << timer->getTime() << " ms." << " GFLOPS " << gpuflops<<std::endl;

    checkCudaErrors(hipMemcpy(h_y_d, d_y, m*sizeof(double),hipMemcpyDeviceToHost));

    double reldiff = 0.0f;
    double diff = 0.0f;

    for (int row = 0; row < m; ++row) {
        double maxabs = std::max(std::abs(h_y[row]),std::abs(h_y_d[row]));
        if (maxabs == 0.0) maxabs=1.0;
        reldiff = std::max(reldiff, std::abs(h_y[row] - h_y_d[row])/maxabs);
        diff = std::max(diff, std::abs(h_y[row] - h_y_d[row]));
    }
    std::cout << "Max diff = " << diff << "  Max rel diff = " << reldiff << std::endl;
// ------------------------------- Cleaning up ------------------------------ //

    delete timer;

    checkCudaErrors(hipFree(d_maxnz));
    checkCudaErrors(hipFree(d_ja));
    checkCudaErrors(hipFree(d_as));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));


    delete[] h_ja;
    delete[] h_as;
    delete[] h_x;
    delete[] h_y;
    delete[] h_y_d;
    return 0;
}
